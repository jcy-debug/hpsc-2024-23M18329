#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <hip/hip_runtime.h>

#define NX 41
#define NY 41
#define NT 500
#define NIT 50
#define DX 2.0 / (NX - 1)
#define DY 2.0 / (NY - 1)
#define DT 0.01
#define RHO 1.0
#define NU 0.02

__global__ void compute_b(float* b, float* u, float* v, double dt, double dx, double dy, double rho) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        float u_diff_x = (u[idx + 1] - u[idx - 1]) / (2.0 * dx);
        float v_diff_y = (v[idx + NX] - v[idx - NX]) / (2.0 * dy);
        float u_diff_y = (u[idx + NX] - u[idx - NX]) / (2.0 * dy);
        float v_diff_x = (v[idx + 1] - v[idx - 1]) / (2.0 * dx);

        b[idx] = rho * (1.0 / dt * (u_diff_x + v_diff_y) -
            u_diff_x * u_diff_x -
            2.0 * u_diff_y * v_diff_x -
            v_diff_y * v_diff_y);
    }
}

__global__ void pressure_poisson(float* p, float* pn, float* b, double dx, double dy) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        p[idx] = ((pn[idx + 1] + pn[idx - 1]) * dy * dy +
            (pn[idx + NX] + pn[idx - NX]) * dx * dx -
            b[idx] * dx * dx * dy * dy) / (2 * (dx * dx + dy * dy));
    }

    // Enforce boundary conditions for pressure
    if (i == NX - 1) {
        p[idx] = p[idx - 1]; // dp/dx = 0 at x = 2
    }
    if (i == 0) {
        p[idx] = p[idx + 1]; // dp/dx = 0 at x = 0
    }
    if (j == NY - 1) {
        p[idx] = 0; // p = 0 at y = 2
    }
    if (j == 0) {
        p[idx] = p[idx + NX]; // dp/dy = 0 at y = 0
    }
}

__global__ void update_velocity(float* u, float* v, float* un, float* vn, float* p, double dt, double dx, double dy, double rho, double nu) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = j * NX + i;

    if (j > 0 && j < NY - 1 && i > 0 && i < NX - 1) {
        float un_diff_x = (un[idx] - un[idx - 1]) / dx;
        float un_diff_y = (un[idx] - un[idx - NX]) / dy;
        float vn_diff_x = (vn[idx] - vn[idx - 1]) / dx;
        float vn_diff_y = (vn[idx] - vn[idx - NX]) / dy;
        float p_diff_x = (p[idx + 1] - p[idx]) / (2 * rho * dx);
        float p_diff_y = (p[idx + NX] - p[idx]) / (2 * rho * dy);
        float un_laplace_x = (un[idx + 1] - 2 * un[idx] + un[idx - 1]) / (dx * dx);
        float un_laplace_y = (un[idx + NX] - 2 * un[idx] + un[idx - NX]) / (dy * dy);
        float vn_laplace_x = (vn[idx + 1] - 2 * vn[idx] + vn[idx - 1]) / (dx * dx);
        float vn_laplace_y = (vn[idx + NX] - 2 * vn[idx] + vn[idx - NX]) / (dy * dy);

        u[idx] = un[idx] - un[idx] * dt * un_diff_x -
            vn[idx] * dt * un_diff_y -
            dt * p_diff_x +
            nu * dt * (un_laplace_x + un_laplace_y);

        v[idx] = vn[idx] - un[idx] * dt * vn_diff_x -
            vn[idx] * dt * vn_diff_y -
            dt * p_diff_y +
            nu * dt * (vn_laplace_x + vn_laplace_y);
    }

    // Enforce boundary conditions for velocity
    if (j == 0 || j == NY - 1 || i == 0 || i == NX - 1) {
        u[idx] = 0;
        v[idx] = 0;
    }

    if (j == NY - 1) {
        u[idx] = 1; // velocity boundary condition at the top (moving lid)
    }
}

int main() {
    // Allocate and initialize host arrays
    float* h_u, * h_v, * h_p, * h_b, * h_un, * h_vn, * h_pn;
    size_t size = NX * NY * sizeof(float);

    h_u = (float*)malloc(size);
    h_v = (float*)malloc(size);
    h_p = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_un = (float*)malloc(size);
    h_vn = (float*)malloc(size);
    h_pn = (float*)malloc(size);

    for (int j = 0; j < NY; j++) {
        for (int i = 0; i < NX; i++) {
            int idx = j * NX + i;
            h_u[idx] = 0.0;
            h_v[idx] = 0.0;
            h_p[idx] = 0.0;
            h_b[idx] = 0.0;
        }
    }

    // Allocate device arrays
    float* d_u, * d_v, * d_p, * d_b, * d_un, * d_vn, * d_pn;
    hipMalloc((void**)&d_u, size);
    hipMalloc((void**)&d_v, size);
    hipMalloc((void**)&d_p, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_un, size);
    hipMalloc((void**)&d_vn, size);
    hipMalloc((void**)&d_pn, size);

    // Copy host arrays to device
    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_p, h_p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Write results to files
    std::ofstream ufile("u.dat");
    std::ofstream vfile("v.dat");
    std::ofstream pfile("p.dat");

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((NX + blockSize.x - 1) / blockSize.x, (NY + blockSize.y - 1) / blockSize.y);

    for (int n = 0; n < NT; n++) {
        compute_b << <gridSize, blockSize >> > (d_b, d_u, d_v, DT, DX, DY, RHO);
        hipDeviceSynchronize();

        for (int it = 0; it < NIT; it++) {
            hipMemcpy(d_pn, d_p, size, hipMemcpyDeviceToDevice);
            pressure_poisson << <gridSize, blockSize >> > (d_p, d_pn, d_b, DX, DY);
            hipDeviceSynchronize();
        }

        hipMemcpy(d_un, d_u, size, hipMemcpyDeviceToDevice);
        hipMemcpy(d_vn, d_v, size, hipMemcpyDeviceToDevice);

        update_velocity << <gridSize, blockSize >> > (d_u, d_v, d_un, d_vn, d_p, DT, DX, DY, RHO, NU);
        hipDeviceSynchronize();

        // Apply boundary conditions on the host
        hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_p, d_p, size, hipMemcpyDeviceToHost);

        for (int j = 0; j < NY; j++) {
            h_u[j * NY + 0] = 0;
            h_u[j * NY + (NX - 1)] = 0;
            h_v[j * NY + 0] = 0;
            h_v[j * NY + (NX - 1)] = 0;
        }
        for (int i = 0; i < NX; i++) {
            h_u[0 * NY + i] = 0;
            h_u[(NX - 1) * NY + i] = 1;
            h_v[0 * NY + i] = 0;
            h_v[(NY - 1) * NY + i] = 0;
        }

        if (n % 10 == 0) {
            for (int j = 0; j < NY; j++)
                for (int i = 0; i < NX; i++)
                    ufile << h_u[j * NY + i] << " ";
            ufile << "\n";
            for (int j = 0; j < NY; j++)
                for (int i = 0; i < NX; i++)
                    vfile << h_v[j * NY + i] << " ";
            vfile << "\n";
            for (int j = 0; j < NY; j++)
                for (int i = 0; i < NX; i++)
                    pfile << h_p[j * NY + i] << " ";
            pfile << "\n";
        }
    }

    ufile.close();
    vfile.close();
    pfile.close();

    // Free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_p);
    hipFree(d_b);
    hipFree(d_un);
    hipFree(d_vn);
    hipFree(d_pn);

    // Free host memory
    free(h_u);
    free(h_v);
    free(h_p);
    free(h_b);
    free(h_un);
    free(h_vn);
    free(h_pn);

    return 0;
}
